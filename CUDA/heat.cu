#include <iostream>
#include <cmath>
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdint>

#define IDX(i, j, ny) ((i) * (ny) + (j))

const float ALPHA = 0.01f;
const float DX = 0.02f;
const float DY = 0.02f;
const float DT = 0.0005f;


void get_color(float value, uint8_t &r, uint8_t &g, uint8_t &b) {
    if (value >= 500.0f)        { r = 255; g = 0;   b = 0;   }
    else if (value >= 100.0f)   { r = 255; g = 128; b = 0;   }
    else if (value >= 50.0f)    { r = 171; g = 71;  b = 188; }
    else if (value >= 25.0f)    { r = 255; g = 255; b = 0;   }
    else if (value >= 1.0f)     { r = 0;   g = 0;   b = 255; }
    else if (value >= 0.1f)     { r = 5;   g = 248; b = 252; }
    else                        { r = 255; g = 255; b = 255; }
}

void write_bmp(const char* filename, float* grid, int width, int height) {
    FILE* f = fopen(filename, "wb");
    if (!f) {
        fprintf(stderr, "Error: Cannot open file %s for writing.\n", filename);
        return;
    }

    const int padding = (4 - (width * 3) % 4) % 4;
    const int file_size = 54 + (3 * width + padding) * height;

    uint8_t header[54] = {
        'B', 'M',               // Signature
        0,0,0,0,                // File size in bytes
        0,0, 0,0,               // Reserved
        54,0,0,0,               // Offset to image data
        40,0,0,0,               // Info header size
        0,0,0,0,                // Width
        0,0,0,0,                // Height
        1,0,                    // Planes
        24,0,                   // Bits per pixel
        0,0,0,0,                // Compression (0 = none)
        0,0,0,0,                // Image size (can be 0 for BI_RGB)
        0,0,0,0, 0,0,0,0,       // X/Y resolution
        0,0,0,0, 0,0,0,0        // Color palette
    };

    // Set width and height
    header[18] =  width        & 0xFF;
    header[19] = (width  >> 8) & 0xFF;
    header[20] = (width  >>16) & 0xFF;
    header[21] = (width  >>24) & 0xFF;
    header[22] =  height        & 0xFF;
    header[23] = (height >> 8) & 0xFF;
    header[24] = (height >>16) & 0xFF;
    header[25] = (height >>24) & 0xFF;

    // Set file size
    header[2] =  file_size        & 0xFF;
    header[3] = (file_size  >> 8) & 0xFF;
    header[4] = (file_size  >>16) & 0xFF;
    header[5] = (file_size  >>24) & 0xFF;

    fwrite(header, 1, 54, f);

    // Write pixel data (bottom to top)
    for (int i = height - 1; i >= 0; --i) {
        for (int j = 0; j < width; ++j) {
            uint8_t r, g, b;
            get_color(grid[i * width + j], r, g, b);
            fwrite(&b, 1, 1, f);
            fwrite(&g, 1, 1, f);
            fwrite(&r, 1, 1, f);
        }
        for (int k = 0; k < padding; ++k)
            fputc(0, f);
    }

    fclose(f);
    printf("BMP image written to %s\n", filename);
}

__global__
void heat_step_kernel(float* current, float* next, int nx, int ny, float r) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i > 0 && i < nx - 1 && j > 0 && j < ny - 1) {
        next[IDX(i,j,ny)] = current[IDX(i,j,ny)]
            + r * (current[IDX(i+1,j,ny)] + current[IDX(i-1,j,ny)] - 2.0f * current[IDX(i,j,ny)])
            + r * (current[IDX(i,j+1,ny)] + current[IDX(i,j-1,ny)] - 2.0f * current[IDX(i,j,ny)]);
    }
}

void initialize(float* grid, int nx, int ny, float temp_source) {
    for (int i = 0; i < nx; i++) {
        for (int j = 0; j < ny; j++) {
            if (i == j || i == ny - j - 1) {
                grid[IDX(i,j,ny)] = temp_source;
            } else {
                grid[IDX(i,j,ny)] = 0.0f;
            }
        }
    }
}

int main(int argc, char** argv) {
    if (argc != 5) {
        std::cerr << "Usage: ./heat_cuda nx ny steps blockSize\n";
        return 1;
    }

    int nx, ny = atoi(argv[1]);
    int steps = atoi(argv[2]);
    int blockSize = atoi(argv[3]);

    float r = ALPHA * DT / (DX * DY);
    size_t size = nx * ny * sizeof(float);

    float* h_grid = (float*)malloc(size);
    float* h_result = (float*)malloc(size);
    initialize(h_grid, nx, ny, 1500.0f);

    float *d_current, *d_next;
    hipMalloc(&d_current, size);
    hipMalloc(&d_next, size);
    hipMemcpy(d_current, h_grid, size, hipMemcpyHostToDevice);

    dim3 block(blockSize, blockSize);
    dim3 grid((ny + block.x - 1) / block.x, (nx + block.y - 1) / block.y);

    for (int t = 0; t < steps; t++) {
        heat_step_kernel<<<grid, block>>>(d_current, d_next, nx, ny, r);
        std::swap(d_current, d_next);
    }

    hipMemcpy(h_result, d_current, size, hipMemcpyDeviceToHost);

    write_bmp("output.bmp", h_result, nx, ny);

    hipFree(d_current);
    hipFree(d_next);
    free(h_grid);
    free(h_result);

    return 0;
}
